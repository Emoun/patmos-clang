#include "hip/hip_runtime.h"
// RUN: %clang_cc1 %s -triple nvptx-- -fcuda-is-device -emit-llvm -o - | FileCheck %s

#include "../SemaCUDA/hip/hip_runtime.h"

#define N 32

extern __shared__ int x;


__global__ void explicit_address_space_cast(int* p) {
	// CHECK: @_Z27explicit_address_space_castPi
   __shared__ unsigned char x[N];

   for (unsigned int i=0; i<(N/4); i++) {
     ((unsigned int *)x)[i] = 0;
		// CHECK: addrspacecast
   }
}

__global__ void pointer_as_array_access() {
	// CHECK: @_Z23pointer_as_array_accessv
   __shared__ int A[10];
   int* p = A + 1;
   p[x] = 0;
	 // CHECK: addrspacecast
}

__device__ int* callee(int* p) {
	// CHECK: @_Z6calleePi
  return p;
}

__global__ void caller() {
	// CHECK: @_Z6callerv
  __shared__ int A[10];
  __shared__ int* p;
	p = A;
	// CHECK: addrspacecast

	((int*)A)[x] = 42;
	// CHECK: addrspacecast
	((int*)A)[0] = 15;
	// CHECK: addrspacecast

  int *np = callee(p);
	A[2] = 5;
	np[0] = 2;
}
